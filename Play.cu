#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "DS_timer.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <windows.h>
#include <conio.h>

#define ROW 32
#define COL 32

#define NONE -1
#define DEAD 0
#define LIVE 1
#define PLAYER 2

#define UP 119 // WŰ
#define LEFT 97 // AŰ
#define DOWN 115 // SŰ
#define RIGHT 100 // DŰ


void initfield(int* _gamefield1, int* _gamefield2, int* Player_Location);
void draw(int* _gamefield);
void Menu();

__global__ void game(int* gamefieldOriginal, int* gamefieldBuffer)
{
	int width = blockDim.x;
	int height = gridDim.x;
	int blockID = threadIdx.x;

	int gridID = blockDim.x * blockIdx.x + blockID;

	if (gamefieldOriginal[gridID] == NONE) {
		gamefieldBuffer[gridID] = NONE;
	}
	else {
		int neighbors = 0;
		if (gamefieldOriginal[gridID - width - 1] == LIVE || gamefieldOriginal[gridID - width - 1] == PLAYER) { // upper left.
			neighbors++;
		}
		if (gamefieldOriginal[gridID - width] == LIVE || gamefieldOriginal[gridID - width] == PLAYER) { // upper.
			neighbors++;
		}
		if (gamefieldOriginal[gridID - width + 1] == LIVE || gamefieldOriginal[gridID - width + 1] == PLAYER) { // upper right.
			neighbors++;
		}
		if (gamefieldOriginal[gridID - 1] == LIVE || gamefieldOriginal[gridID - 1] == PLAYER) { // left.
			neighbors++;
		}
		if (gamefieldOriginal[gridID + 1] == LIVE || gamefieldOriginal[gridID + 1] == PLAYER) { // right.
			neighbors++;
		}
		if (gamefieldOriginal[gridID + width - 1] == LIVE || gamefieldOriginal[gridID + width - 1] == PLAYER) { // lower left.
			neighbors++;
		}
		if (gamefieldOriginal[gridID + width] == LIVE || gamefieldOriginal[gridID + width] == PLAYER) { // lower.
			neighbors++;
		}
		if (gamefieldOriginal[gridID + width + 1] == LIVE || gamefieldOriginal[gridID + width + 1] == PLAYER) { // lower right.
			neighbors++;
		}

		if (gamefieldOriginal[gridID] == DEAD) {
			if (neighbors == 3) {
				gamefieldBuffer[gridID] = LIVE;
			}
		}
		else if (gamefieldOriginal[gridID] == LIVE) {
			if (neighbors < 2 || neighbors > 3) {
				gamefieldBuffer[gridID] = DEAD;
			}
		}
	}

}

__global__ void copy(int* gamefieldOriginal, int* gamefieldBuffer) {
	int width = blockDim.x;
	int height = gridDim.x;
	int blockID = threadIdx.x;

	int gridID = blockDim.x * blockIdx.x + blockID;
	gamefieldOriginal[gridID] = gamefieldBuffer[gridID];
}

int main()
{
	srand(time(NULL));

	int width = COL;
	int height = ROW;
	int size = sizeof(int) * width * height;

	int term = 30;
	int count = 0;

	int* Player_Location = new int;
	*Player_Location = COL + 1; // �÷��̾��� ���� ���� �������ִ� ��
	int eat = 0; // ��Ƹ��� ���� ��
	

	int* gamefieldParallelHost;
	int* gamefieldParallelCUDA;
	int* gamefieldBufferCUDA;
	int* gamefieldPlayer; // �÷��̾��� ��ġ�� ����ٰ� �ٲ㼭 �ٲ� ����� ����̽� �޸𸮿� �����Ұ���

	hipMalloc(&gamefieldParallelCUDA, size);
	hipMalloc(&gamefieldBufferCUDA, size);

	gamefieldParallelHost = new int[width * height];
	gamefieldPlayer = new int[width * height];

	memset(gamefieldParallelHost, 0, size);
	memset(gamefieldPlayer, 0, size);

	initfield(gamefieldParallelHost, gamefieldPlayer, Player_Location);

	dim3 dimBlock(width);
	dim3 dimGrid(height);

	hipMemcpy(gamefieldBufferCUDA, gamefieldParallelHost, size, hipMemcpyHostToDevice);
	hipMemcpy(gamefieldParallelCUDA, gamefieldParallelHost, size, hipMemcpyHostToDevice);

	char key = '\0'; // Ű���� �Է� ���� �� �ʱ�ȭ

	Menu();

	while (count < term)
	{
		// ������ �����ϱ����� ���� Ű���� �Է��� �޾ƾ� count�� ���� �����ϸ� ������ ���۵ȴ�
		// W : ���� �̵� A: �������� �̵� S: �Ʒ��� �̵� D : ���������� �̵�
		// �� �����δ� ������ ���Ѵ�
		// term �ð� ���� ��Ƹ��� ���� ���� �� ����

		hipMemcpy(gamefieldPlayer, gamefieldParallelCUDA, size, hipMemcpyDeviceToHost);

		if (_kbhit()) // Ű �Է¹����� true �����ϴ� �Լ�
		{
			// �÷��̾��� ��ġ�� Player_temp ������ ����
			int Player_temp = *Player_Location;

			key = _getch(); // �Է¹��� Ű ���� key ������ ����

			if (key == UP) // WŰ �Է½� ���� �̵�
			{
				Player_temp = Player_temp - COL;
			}
			else if (key == LEFT) // AŰ �Է½� ���� �̵�
			{
				Player_temp = Player_temp - 1;
			}
			else if (key == DOWN) // SŰ �Է½� �Ʒ��� �̵�
			{
				Player_temp = Player_temp + COL;
			}
			else if (key == RIGHT) // DŰ �Է½� ������ �̵�
			{
				Player_temp = Player_temp + 1;
			}
			else
			{
				printf("�߸� �Է��߽��ϴ�!!!!!\n");
			}

			// �̵��ϰ��� �ϴ� ���� �� �� ������ �̵��� �Ұ�
			if (gamefieldPlayer[Player_temp] == NONE)
				printf("���̶� ������!!!!!!\n");
			else
			{
				if (gamefieldPlayer[Player_temp] == LIVE) // �̵��� ���� ������ ��������� ����
					eat++;

				gamefieldPlayer[*Player_Location] = DEAD; // �̵��ϱ����� ���� �ִ� ���� DEAD ���·� �ٲ���
				*Player_Location = Player_temp; 
				gamefieldPlayer[*Player_Location] = PLAYER; // �̵��� ���� ���ο� �÷��̾��� ��ġ�� ����

				// �ٲ� gamefieldPlayer�� Ŀ�� �޸𸮿� ��������
				hipMemcpy(gamefieldParallelCUDA, gamefieldPlayer, size, hipMemcpyHostToDevice); 
				hipMemcpy(gamefieldBufferCUDA, gamefieldPlayer, size, hipMemcpyHostToDevice);
			}
		}

		// 1�ʸ��� Ŀ�� �Լ����� �����(1�ʸ��� ���� �׷���)
		game << <dimGrid, dimBlock >> > (gamefieldParallelCUDA, gamefieldBufferCUDA);
		copy << <dimGrid, dimBlock >> > (gamefieldParallelCUDA, gamefieldBufferCUDA);

		hipDeviceSynchronize();
		hipMemcpy(gamefieldParallelHost, gamefieldParallelCUDA, size, hipMemcpyDeviceToHost);

		draw(gamefieldParallelHost);
		printf("���� ���� ���� �� : %d\n", eat);
		printf("%d�� ���ҽ��ϴ�\n", term - count);
		Sleep(1000);
		count++;
		system("cls");
	}
	
	printf("�� ���� ���� : %d\n", eat);

	hipFree(gamefieldParallelCUDA);
	hipFree(gamefieldBufferCUDA);

	delete[] gamefieldPlayer; delete[] gamefieldParallelHost;
	return 0;
}

void initfield(int* _gamefield1, int* _gamefield2 ,int* _Player_Location)
{
	for (int i = 0; i < ROW * COL; i++)
		_gamefield1[i] = rand() % 2;

	for (int i = 0; i < COL; i++)
	{
		_gamefield1[i] = NONE; // �� ��
		_gamefield1[i + COL * (ROW - 1)] = NONE; // �� �Ʒ�
	}

	for (int i = 0; i < ROW; i++)
	{
		_gamefield1[COL * i] = NONE; // �� ����
		_gamefield1[COL * (i + 1) - 1] = NONE; // �� ������
	}

	_gamefield1[*_Player_Location] = PLAYER;

	for (int i = 0; i < ROW * COL; i++) {
		_gamefield2[i] = _gamefield1[i];
	}
}

void draw(int* _gamefield)
{
	for (int i = 0; i < ROW; i++)
	{
		for (int j = 0; j < COL; j++)
		{
			printf("[%2d]", _gamefield[i * ROW + j]);
		}
		printf("\n");
	}
}

void Menu()
{
	printf("-----------------------------------------------------------------------\n");
	printf("\t\t\t���� �Ա� �����Դϴ�\n");
	printf("�÷��̾��� ��ŸƮ�� �� ���� �� ��ġ���� �����մϴ�(WASD�� �̵��غ�����)\n");
	printf("\t\t(����) �����δ� ���� ���մϴ�\n");
	printf("���� �ð����� ������ �󸶳� ���� ���� �� �ִ��� �����غ�����\n");
	printf("\t\t�������� ����ؼ� �װų� �����˴ϴ�\n");
	printf("-----------------------------------------------------------------------\n");

	for (int i = 0; i < 10; i++)
	{
		printf("%d�� �Ŀ� ���۵˴ϴ�\n", 10 - i);
		Sleep(1000);
	}
	system("cls");
}